// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N (1 << 22)

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

void multiplyCPU(const float *h_v1, const float *h_v2, float *h_out) {
    for (int i = 0; i < N; i++) {
        h_out[i] = h_v1[i] * h_v2[i];
    }
}

void expensiveFunctionCPU(const float *h_v1, const float *h_v2, float *h_out) {
    for (int i = 0; i < N; i++) {
        float a = h_v1[i], b = h_v2[i];
        h_out[i] = (a * b) * (sqrt(a + b) + sqrt(a) + sqrt(b - a) + sqrt(b));
    }
}

__global__ void multiplyGPU(const float *g_v1, const float *g_v2, float *g_out) {
    // YOUR CODE GOES HERE
    // Implement GPU version of multiplyCPU
    // (a)

}

__global__ void expensiveFunctionGPU(const float *g_v1, const float *g_v2, float *g_out) {
    // YOUR CODE GOES HERE
    // Implement GPU version of expensiveFunctionCPU
    // (c)
    
}

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) {
    ///////////////////// LOTS OF INITIALIZATION CODE ////////////////////////
    // pointers to host vectors v1 and v1
    float *h_v1, *h_v2;
    // pointers to host output for multiply and expensive
    float *h_multiply_correct, *h_expensive_correct;
    // pointers to store results from gpu functions on the host
    float *h_multiply_out, *h_expensive_out;

    // pointer for device vectors v1 and v2
    float *d_v1, *d_v2;
    // pointers to device output
    float *d_multiply_out, *d_expensive_out;

    

    // Number of test iterations to use for timing
    int testIterations = 3;

    // allocate memory for pointers
    h_v1 = (float *)malloc(N * sizeof(float));
    h_v2 = (float *)malloc(N * sizeof(float));
    h_multiply_correct  = (float *)malloc(N * sizeof(float));
    h_expensive_correct = (float *)malloc(N * sizeof(float));
    h_multiply_out  = (float *)malloc(N * sizeof(float));
    h_expensive_out = (float *)malloc(N * sizeof(float));

    // allocate memory for device pointers
    hipMalloc( (void **) &d_v1, N * sizeof(float));
    hipMalloc( (void **) &d_v2, N * sizeof(float));
    hipMalloc( (void **) &d_multiply_out, N *sizeof(float));
    hipMalloc( (void **) &d_expensive_out, N *sizeof(float));

    // Initialize v1 and v2
    for (int i = 0; i < N; i++) {
        h_v1[i] = 1.0 * (i % 10000);
        h_v2[i] = 2 * h_v1[i];
    }

    // CPU functions over testIterations
    for (int i = 0; i < testIterations; i++) {
        // zero memory for outputs
        memset(h_multiply_correct,  0, N * sizeof(float));
        memset(h_expensive_correct,  0, N * sizeof(float));
        
				// run cpu kernel
        multiplyCPU(h_v1, h_v2, h_multiply_correct);
        
        // run cpu kernel
        expensiveFunctionCPU(h_v1, h_v2, h_expensive_correct);
    }

    //////////////////////// INSERT CODE IN THIS SECTION /////////////////////
    // GPU functions over testIterations
    for (int i = 0; i < testIterations; i++) {
        // zero output memory
        memset(h_multiply_out, 0, N * sizeof(float));
        memset(h_expensive_out, 0, N * sizeof(float));
        hipMemset(d_multiply_out, 0, N * sizeof(float));
        hipMemset(d_expensive_out, 0, N * sizeof(float));
        // zero input memory
        hipMemset(d_v1, 0, N * sizeof(float));
        hipMemset(d_v2, 0, N * sizeof(float));
        
        // transfer data to GPU
        hipMemcpy(d_v1, h_v1, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_v2, h_v2, N * sizeof(float), hipMemcpyHostToDevice);

        // YOUR CODE GOES HERE
        // Execute multiply kernel
        // (b)
       
        // transfer data from GPU
        hipMemcpy(h_multiply_out, d_multiply_out, N * sizeof(float), hipMemcpyDeviceToHost);


        // Check for any CUDA errors
        checkCUDAError("multiplyGPU");

        // zero input memory
        hipMemset(d_v1, 0, N * sizeof(float));
        hipMemset(d_v2, 0, N * sizeof(float));

        // transfer data to GPU
        hipMemcpy(d_v1, h_v1, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_v2, h_v2, N * sizeof(float), hipMemcpyHostToDevice);

        // YOUR CODE GOES HERE
        // Execute expensive function kernel
        // (d)
        

        // transfer data from GPU
        hipMemcpy(h_expensive_out, d_expensive_out, N * sizeof(float), hipMemcpyDeviceToHost);

        // Check for any CUDA errors
        checkCUDAError("expensiveFunctionGPU");
    }

    
    /////////////////////////////// VALIDATION ///////////////////////////////

    // check if output from gpu kernels is correct
    for (int i = 0; i < N; i++) {
        if (!(abs(h_multiply_out[i] - h_multiply_correct[i]) <= 0.0001)) {
            printf("Test failed (h_multiply_out[%d]:%f != h_multiply_correct[%d]:%f)!\n", 
                   i, h_multiply_out[i], i, h_multiply_correct[i]);
            exit(1);
        }
        if (!(abs(h_expensive_out[i] - h_expensive_correct[i]) <= 0.00001 * abs(h_expensive_correct[i]))) {
            printf("Test failed (h_expensive_out[%d]:%f != h_expensive_correct[%d]:%f)!\n", 
                   i, h_expensive_out[i], i, h_expensive_correct[i]);
            exit(1);
        }
    }

    printf("Test passed!\n");

    //////////////////////////////// CLEANUP /////////////////////////////////
    // free host memory
    free(h_v1);
    free(h_v2);
    free(h_multiply_correct);
    free(h_expensive_correct);
    free(h_multiply_out);
    free(h_expensive_out);

    // free device memory
    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(d_multiply_out);
    hipFree(d_expensive_out);
    return 0;
}
