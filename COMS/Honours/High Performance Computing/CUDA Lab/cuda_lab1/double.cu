
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  /*
   * The size of this grid is 256*32 = 8192.
   */

  //In order to ensure that everything works, we need to have that
  //that threads_per_block*number_of_blocks > N
  size_t threads_per_block = 256;
  size_t number_of_blocks = 64;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
