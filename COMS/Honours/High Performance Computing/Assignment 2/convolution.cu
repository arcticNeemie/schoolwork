/*
 *
 * This program takes an input grayscale image and applies a sepcified filter
 * using image convolution
 */

// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include "helper_functions.h"    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include "hip/hip_runtime_api.h"         // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f

//Image files
const char *imageFilename = "lena_bw.pgm";

//Functions
void printImage(float* hData, int width, int height);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("Starting execution\n");
    //Load Image
    printf("Loading image: %s\n",imageFilename);
    float *hData = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename, argv[0]);
    if (imagePath == NULL){
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }
    sdkLoadPGM(imagePath, &hData, &width, &height);
    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);
    
}

////////////////////////////////////////////////////////////////////////////////
// Utility Functions
////////////////////////////////////////////////////////////////////////////////
void printImage(float* hData, int width, int height){
    for(int i=0;i<height;i++){
        for(int j=0;j<width;j++){
            printf("%f",hData[i*height+j]);
        }
        printf("\n");
    }
}
