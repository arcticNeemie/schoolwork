#include "hip/hip_runtime.h"
/*
 *
 * This program takes an input grayscale image and applies a sepcified filter
 * using image convolution
 */

// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include "helper_functions.h"    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include "hip/hip_runtime_api.h"         // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f
#define FILTERSIZE 3

//Image files
const char *sobelName = "_sobel_";
const char *sharpenName = "_sharpen_";
const char *averageName = "_average_";

//Functions
void printImage(float* hData, int width, int height);
void saveImage(float* dData,char* imagePath,const char* filter,int width, int height, float time);
void convolveCPU(float* dData, float*hData, float* filter, int width, int height);
void applySerialConvolution(float* hData, float* filter, char* imagePath, const char* name, int width, int height, unsigned int size);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    const char* imageFilename;
    if(argc>1){
      imageFilename = argv[1];
    }
    else{
      imageFilename = "lena_bw.pgm";
    }
    printf("Starting execution\n");
    //Load Image
    printf("Loading image: %s\n",imageFilename);
    float *hData = NULL; //Input
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename, argv[0]);
    if (imagePath == NULL){
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }
    sdkLoadPGM(imagePath, &hData, &width, &height);
    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n\n", imageFilename, width, height);

    //Define Filter
    float averagingFilter[] = {1.0/9,1.0/9,1.0/9,1.0/9,1.0/9,1.0/9,1.0/9,1.0/9,1.0/9}; //Averaging Filter
    float sharpeningFilter[] = {-1,-1,-1,-1,9,-1,-1,-1,-1}; //Sharpening Filter
    float sobelFilter[] = {-1,0,1,-2,0,2,-1,0,1}; //Sobel Filter

    //Apply serial convolution
    printf("Beginning serial convolution...\n");
    applySerialConvolution(hData,averagingFilter,imagePath,averageName,width,height,size);
    applySerialConvolution(hData,sharpeningFilter,imagePath,sharpenName,width,height,size);
    applySerialConvolution(hData,sobelFilter,imagePath,sobelName,width,height,size);
    printf("Finished serial convolution!\n\n");

    //Apply naive parallelization implementation
    //TODO

    //Apply shared memory implementation
    //TODO

    //Apply constant memory implementation
    //TODO

    //Apply texture memory implementation
    //TODO

}

////////////////////////////////////////////////////////////////////////////////
// Convolutions
////////////////////////////////////////////////////////////////////////////////

//Serial convolution
void convolveCPU(float* dData, float*hData, float* filter, int width, int height){
  float sum;
  int adjust = FILTERSIZE/2; //Integer division should floor
  int x1, y1;
  for(int x=0;x<height;x++){
    for(int y=0;y<width;y++){
      sum = 0;
      for(int s=0;s<FILTERSIZE;s++){
        for(int t=0;t<FILTERSIZE;t++){
          x1 = x-s+adjust;
          y1 = y-t+adjust;
          if(x1>=0 && x1<height && y1>=0 && y1<width){
              sum += hData[x1*width+y1]*filter[s*FILTERSIZE+t];
          }
        }
      }
      dData[x*width+y] = sum;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Utility Functions
////////////////////////////////////////////////////////////////////////////////

//Print out the image as a matrix
void printImage(float* hData, int width, int height){
    for(int i=0;i<height;i++){
        for(int j=0;j<width;j++){
            printf("%f",hData[i*height+j]);
        }
        printf("\n");
    }
}

//Save image to file
void saveImage(float* dData,char* imagePath,const char* filter,int width, int height, float time){
  char outputFilename[1024];
  char* sub = (char*) malloc(strlen(filter)+strlen("out"));
  strcpy(sub,filter);
  strcat(sub,"serial_out");
  int offset = strlen(imagePath)/sizeof(char) - 4;
  strncpy(outputFilename,imagePath,offset);
  outputFilename[offset] = '\0';
  strcat(outputFilename,sub);
  strcat(outputFilename,imagePath+offset);
  sdkSavePGM(outputFilename, dData, width, height);
  printf("Convolved in serial in %f s, saved to '%s'\n", time, outputFilename);
}

//Apply a filter in serial, time it and save result
void applySerialConvolution(float* hData, float* filter, char* imagePath, const char* name, int width, int height, unsigned int size){
  float *dData = (float*) malloc(size); //Output
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);
  convolveCPU(dData,hData,filter,width,height);
  sdkStopTimer(&timer);
  float time = sdkGetTimerValue(&timer) / 1000.0f;
  sdkDeleteTimer(&timer);
  saveImage(dData,imagePath,name,width,height,time);
}
