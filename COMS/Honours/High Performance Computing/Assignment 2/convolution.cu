#include "hip/hip_runtime.h"
/*
 *
 * This program takes an input grayscale image and applies a sepcified filter
 * using image convolution
 */

// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include "helper_functions.h"    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include "hip/hip_runtime_api.h"         // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f

//Image files
const char *sobelName = "_sobel_";
const char *sharpenName = "_sharpen_";
const char *averageName = "_average_";

//Functions
void printImage(float* hData, int width, int height);
void saveImage(float* dData,char* imagePath,const char* filter,
    const char* type, int width, int height, float time);

void convolveCPU(float* dData, float*hData, float* filter,
    int width, int height, int filtersize);

void applySerialConvolution(float* hData, float* filter, char* imagePath,
    const char* name, int width, int height, unsigned int size, int filtersize);
void applyNaiveParallelConvolution(float* hData, float* filter, char* imagePath,
    const char* name, int width, int height, unsigned int size, int filtersize);

////////////////////////////////////////////////////////////////////////////////
// Convolutions
////////////////////////////////////////////////////////////////////////////////

//Serial convolution
void convolveCPU(float* dData, float*hData, float* filter, int width, int height, int filtersize){
  float sum;
  int adjust = filtersize/2; //Integer division should floor
  int x1, y1;
  for(int x=0;x<height;x++){
    for(int y=0;y<width;y++){
      sum = 0;
      for(int s=0;s<filtersize;s++){
        for(int t=0;t<filtersize;t++){
          x1 = x-s+adjust;
          y1 = y-t+adjust;
          if(x1>=0 && x1<height && y1>=0 && y1<width){
              sum += hData[x1*width+y1]*filter[s*filtersize+t];
          }
        }
      }
      if(sum>1){
          sum = 1;
      }
      else if(sum<0){
          sum = 0;
      }
      dData[x*width+y] = sum;
    }
  }
}

__global__ void convolveGPUNaive(float* dData,float* hData,float* filter,int width,int height, int filtersize){
  unsigned int x = threadIdx.x + blockDim.x*blockIdx.x;
  unsigned int y = threadIdx.y + blockDim.y*blockIdx.y;

  int adjust = filtersize/2;
  int x1,y1;
  if(x<height && y<width){
    float sum = 0;
    for(int s=0;s<filtersize;s++){
      for(int t=0;t<filtersize;t++){
        x1 = x-s+adjust;
        y1 = y-t+adjust;
        if(x1>=0 && x1<height && y1>=0 && y1<width){
          if(x1*width+y1<width*height && s*filtersize+t<filtersize*filtersize){
            sum += hData[x1*width+y1]*filter[s*filtersize+t];
          }
          else{
            printf("Hello\n");
          }

        }
      }
    }
    if(sum>1){
        sum = 1;
    }
    else if(sum<0){
        sum = 0;
    }
    dData[x*width+y] = sum;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    const char* imageFilename;
    if(argc>1){
      imageFilename = argv[1];
    }
    else{
      imageFilename = "lena_bw.pgm";
    }
    printf("Starting execution\n");
    //Load Image
    printf("Loading image: %s\n",imageFilename);
    float *hData = NULL; //Input
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename, argv[0]);
    if (imagePath == NULL){
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }
    sdkLoadPGM(imagePath, &hData, &width, &height);
    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n\n", imageFilename, width, height);

    //Define Filter
    int filtersize = 3;
    float averagingFilter[] = {1.0/9,1.0/9,1.0/9,1.0/9,1.0/9,1.0/9,1.0/9,1.0/9,1.0/9}; //Averaging Filter
    float sharpeningFilter[] = {-1,-1,-1,-1,9,-1,-1,-1,-1}; //Sharpening Filter
    float sobelFilter[] = {-1,0,1,-2,0,2,-1,0,1}; //Sobel Filter

    //Apply serial convolution
    printf("Beginning serial convolution...\n");
    applySerialConvolution(hData,averagingFilter,imagePath,averageName,width,height,size,filtersize);
    applySerialConvolution(hData,sharpeningFilter,imagePath,sharpenName,width,height,size,filtersize);
    applySerialConvolution(hData,sobelFilter,imagePath,sobelName,width,height,size,filtersize);
    printf("Finished serial convolution!\n\n");

    //Apply naive parallelization implementation
    //TODO
    printf("Beginning naive parallel convolution...\n");
    applyNaiveParallelConvolution(hData,averagingFilter,imagePath,averageName,width,height,size,filtersize);

    //Apply shared memory implementation
    //TODO

    //Apply constant memory implementation
    //TODO

    //Apply texture memory implementation
    //TODO

    //Free
    free(imagePath);

}

////////////////////////////////////////////////////////////////////////////////
// Utility Functions
////////////////////////////////////////////////////////////////////////////////

//Print out the image as a matrix
void printImage(float* hData, int width, int height){
    for(int i=0;i<height;i++){
        for(int j=0;j<width;j++){
            printf("%f",hData[i*height+j]);
        }
        printf("\n");
    }
}

//Save image to file
void saveImage(float* dData,char* imagePath,const char* filter, const char* type, int width, int height, float time){
  char outputFilename[1024];
  char* sub = (char*) malloc(strlen(filter)+strlen("out"));
  strcpy(sub,filter);
  strcat(sub,type);
  strcat(sub,"_out");
  int offset = strlen(imagePath)/sizeof(char) - 4;
  strncpy(outputFilename,imagePath,offset);
  outputFilename[offset] = '\0';
  strcat(outputFilename,sub);
  strcat(outputFilename,imagePath+offset);
  sdkSavePGM(outputFilename, dData, width, height);
  printf("Convolved in serial in %f s, saved to '%s'\n", time, outputFilename);
  free(sub);
}

////////////////////////////////////////////////////////////////////////////////
// Application Functions
////////////////////////////////////////////////////////////////////////////////

//Apply a filter in serial, time it and save result
void applySerialConvolution(float* hData, float* filter, char* imagePath, const char* name, int width, int height, unsigned int size, int filtersize){
  const char* type = "serial";
  float *dData = (float*) malloc(size); //Output
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);
  convolveCPU(dData,hData,filter,width,height,filtersize);
  sdkStopTimer(&timer);
  float time = sdkGetTimerValue(&timer) / 1000.0f;
  sdkDeleteTimer(&timer);
  saveImage(dData,imagePath,name,type,width,height,time);
  free(dData);
}

//Apply a filter in the naive parallel approach, time it and save result
void applyNaiveParallelConvolution(float* hData, float* filter, char* imagePath, const char* name, int width, int height, unsigned int size, int filtersize){
  //int devID = findCudaDevice(argc, (const char **) argv);
  // Allocate device memory for result
  float *dData = NULL;
  // Allocate device memory and copy image data
  checkCudaErrors(hipMalloc((void **) &dData, size));
  checkCudaErrors(hipMemcpy(dData,hData,size,hipMemcpyHostToDevice));

  int fsize = filtersize*filtersize*sizeof(float);
  float *dFilter = NULL;
  checkCudaErrors(hipMalloc((void **) &dFilter, fsize));
  checkCudaErrors(hipMemcpy(dFilter,filter,fsize,hipMemcpyHostToDevice));

  dim3 dimBlock(8, 8, 1);
  dim3 dimGrid(height / dimBlock.x, width / dimBlock.y, 1);
  checkCudaErrors(hipDeviceSynchronize());
  //Time
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);
  //Execute kernel
  convolveGPUNaive<<<dimGrid, dimBlock,0>>>(dData,hData,dFilter,width,height,filtersize);
  // Check if kernel execution generated an error
  getLastCudaError("Kernel execution failed");

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  float time = sdkGetTimerValue(&timer)/1000.0f;
  const char* type = "naive";

  // Allocate mem for the result on host side
  float *hOutputData = (float *) malloc(size);
  //Copy result from device to host
  checkCudaErrors(hipMemcpy(hOutputData,dData,size,hipMemcpyDeviceToHost));

  saveImage(hOutputData,imagePath,name,type,width,height,time);
  sdkDeleteTimer(&timer);

  hipDeviceReset();
}
